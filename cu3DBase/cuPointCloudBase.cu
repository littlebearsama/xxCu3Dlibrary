#include "hip/hip_runtime.h"
#include"cuPointCloudBase.h"
#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime.h"
#include "hip/hip_texture_types.h"


//�����㣬��flags��ֵΪflag_value�ĵ㱣��������������ΪNAN����
__global__ void cuGetSubPts(int N, glm::vec3* pts_in_dev, glm::vec3* pts_out_dev, int* flags, int flag_value)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < N)
	{
		if (flags[index] != flag_value)
		{
			pts_out_dev[index] = glm::vec3(NAN, NAN, NAN);
		}
		else
		{
			pts_out_dev[index] = pts_in_dev[index];
		}
	}
}
