#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "libsgm.h"
#include "internal.h"
#include "utility.hpp"

namespace {
	__global__ void correct_disparity_range_kernel(uint16_t* d_disp, int width, int height, int pitch, int min_disp_scaled, int invalid_disp_scaled) {
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= width || y >= height) {
			return;
		}

		uint16_t d = d_disp[y * pitch + x];
		if (d == sgm::INVALID_DISP) {
			d = invalid_disp_scaled;
		} else {
			d += min_disp_scaled;
		}
		d_disp[y * pitch + x] = d;
	}
}

namespace sgm {
	namespace details {
		void correct_disparity_range(uint16_t* d_disp, int width, int height, int pitch, bool subpixel, int min_disp) {
			if (!subpixel && min_disp == 0) {
				return;
			}

			static constexpr int SIZE = 16;
			const dim3 blocks((width + SIZE - 1) / SIZE, (height + SIZE - 1) / SIZE);
			const dim3 threads(SIZE, SIZE);

			const int scale = subpixel ? StereoSGM::SUBPIXEL_SCALE : 1;
			const int     min_disp_scaled =  min_disp      * scale;
			const int invalid_disp_scaled = (min_disp - 1) * scale;

			correct_disparity_range_kernel<<<blocks, threads>>>(d_disp, width, height, pitch, min_disp_scaled, invalid_disp_scaled);
		}
	}
}
