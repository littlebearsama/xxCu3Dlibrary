/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "sgm.hpp"
#include "census_transform.hpp"
#include "path_aggregation.hpp"
#include "winner_takes_all.hpp"

namespace sgm {

template <typename T, size_t MAX_DISPARITY>
class SemiGlobalMatching<T, MAX_DISPARITY>::Impl {

private:
	CensusTransform<T> m_census_left;
	CensusTransform<T> m_census_right;
	PathAggregation<MAX_DISPARITY> m_path_aggregation;
	WinnerTakesAll<MAX_DISPARITY> m_winner_takes_all;

public:
	Impl()
		: m_census_left()
		, m_census_right()
		, m_path_aggregation()
		, m_winner_takes_all()
	{ }

	void enqueue(
		output_type *dest_left,
		output_type *dest_right,
		const input_type *src_left,
		const input_type *src_right,
		int width,
		int height,
		int src_pitch,
		int dst_pitch,
		const StereoSGM::Parameters& param,
		hipStream_t stream)
	{
		m_census_left.enqueue(
			src_left, width, height, src_pitch, stream);
		m_census_right.enqueue(
			src_right, width, height, src_pitch, stream);
		m_path_aggregation.enqueue(
			m_census_left.get_output(),
			m_census_right.get_output(),
			width, height,
			param.path_type, param.P1, param.P2, param.min_disp,
			stream);
		m_winner_takes_all.enqueue(
			dest_left, dest_right,
			m_path_aggregation.get_output(),
			width, height, dst_pitch,
			param.uniqueness, param.subpixel, param.path_type,
			stream);
	}

};


template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::SemiGlobalMatching()
	: m_impl(new Impl())
{ }

template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::~SemiGlobalMatching() = default;


template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::execute(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		0);
	hipStreamSynchronize(0);
}

template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::enqueue(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param,
	hipStream_t stream)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		stream);
}


template class SemiGlobalMatching<uint8_t,   64>;
template class SemiGlobalMatching<uint8_t,  128>;
template class SemiGlobalMatching<uint8_t,  256>;
template class SemiGlobalMatching<uint16_t,  64>;
template class SemiGlobalMatching<uint16_t, 128>;
template class SemiGlobalMatching<uint16_t, 256>;

}
