#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <cstdio>
#include "oblique_path_aggregation.hpp"
#include "path_aggregation_common.hpp"

namespace sgm {
namespace path_aggregation {

static constexpr unsigned int DP_BLOCK_SIZE = 16u;
static constexpr unsigned int BLOCK_SIZE = WARP_SIZE * 8u;

template <int X_DIRECTION, int Y_DIRECTION, unsigned int MAX_DISPARITY>
__global__ void aggregate_oblique_path_kernel(
	uint8_t *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp)
{
	static const unsigned int SUBGROUP_SIZE = MAX_DISPARITY / DP_BLOCK_SIZE;
	static const unsigned int PATHS_PER_WARP = WARP_SIZE / SUBGROUP_SIZE;
	static const unsigned int PATHS_PER_BLOCK = BLOCK_SIZE / SUBGROUP_SIZE;

	static const unsigned int RIGHT_BUFFER_SIZE = MAX_DISPARITY + PATHS_PER_BLOCK;
	static const unsigned int RIGHT_BUFFER_ROWS = RIGHT_BUFFER_SIZE / DP_BLOCK_SIZE;

	static_assert(X_DIRECTION == 1 || X_DIRECTION == -1, "");
	static_assert(Y_DIRECTION == 1 || Y_DIRECTION == -1, "");
	if(width == 0 || height == 0){
		return;
	}

	__shared__ feature_type right_buffer[2 * DP_BLOCK_SIZE][RIGHT_BUFFER_ROWS];
	DynamicProgramming<DP_BLOCK_SIZE, SUBGROUP_SIZE> dp;

	const unsigned int warp_id  = threadIdx.x / WARP_SIZE;
	const unsigned int group_id = threadIdx.x % WARP_SIZE / SUBGROUP_SIZE;
	const unsigned int lane_id  = threadIdx.x % SUBGROUP_SIZE;
	const unsigned int shfl_mask =
		generate_mask<SUBGROUP_SIZE>() << (group_id * SUBGROUP_SIZE);

	const int x0 =
		blockIdx.x * PATHS_PER_BLOCK +
		warp_id * PATHS_PER_WARP +
		group_id +
		(X_DIRECTION > 0 ? -static_cast<int>(height - 1) : 0);
	const int right_x00 =
		blockIdx.x * PATHS_PER_BLOCK +
		(X_DIRECTION > 0 ? -static_cast<int>(height - 1) : 0);
	const unsigned int dp_offset = lane_id * DP_BLOCK_SIZE;

	const unsigned int right0_addr =
		static_cast<unsigned int>(right_x00 + PATHS_PER_BLOCK - 1 - x0) + dp_offset;
	const unsigned int right0_addr_lo = right0_addr % DP_BLOCK_SIZE;
	const unsigned int right0_addr_hi = right0_addr / DP_BLOCK_SIZE;

	for(unsigned int iter = 0; iter < height; ++iter){
		const int y = static_cast<int>(Y_DIRECTION > 0 ? iter : height - 1 - iter);
		const int x = x0 + static_cast<int>(iter) * X_DIRECTION;
		const int right_x0 = right_x00 + static_cast<int>(iter) * X_DIRECTION;
		// Load right to smem
		for(unsigned int i0 = 0; i0 < RIGHT_BUFFER_SIZE; i0 += BLOCK_SIZE){
			const unsigned int i = i0 + threadIdx.x;
			if(i < RIGHT_BUFFER_SIZE){
				const int right_x = static_cast<int>(right_x0 + PATHS_PER_BLOCK - 1 - i - min_disp);
				feature_type right_value = 0;
				if(0 <= right_x && right_x < static_cast<int>(width)){
					right_value = right[right_x + y * width];
				}
				const unsigned int lo = i % DP_BLOCK_SIZE;
				const unsigned int hi = i / DP_BLOCK_SIZE;
				right_buffer[lo][hi] = right_value;
				if(hi > 0){
					right_buffer[lo + DP_BLOCK_SIZE][hi - 1] = right_value;
				}
			}
		}
		__syncthreads();
		// Compute
		if(0 <= x && x < static_cast<int>(width)){
			const feature_type left_value = __ldg(&left[x + y * width]);
			feature_type right_values[DP_BLOCK_SIZE];
			for(unsigned int j = 0; j < DP_BLOCK_SIZE; ++j){
				right_values[j] = right_buffer[right0_addr_lo + j][right0_addr_hi];
			}
			uint32_t local_costs[DP_BLOCK_SIZE];
			for(unsigned int j = 0; j < DP_BLOCK_SIZE; ++j){
				local_costs[j] = __popc(left_value ^ right_values[j]);
			}
			dp.update(local_costs, p1, p2, shfl_mask);
			store_uint8_vector<DP_BLOCK_SIZE>(
				&dest[dp_offset + x * MAX_DISPARITY + y * MAX_DISPARITY * width],
				dp.dp);
		}
		__syncthreads();
	}
}


template <unsigned int MAX_DISPARITY>
void enqueue_aggregate_upleft2downright_path(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	static const unsigned int SUBGROUP_SIZE = MAX_DISPARITY / DP_BLOCK_SIZE;
	static const unsigned int PATHS_PER_BLOCK = BLOCK_SIZE / SUBGROUP_SIZE;

	const int gdim = (width + height + PATHS_PER_BLOCK - 2) / PATHS_PER_BLOCK;
	const int bdim = BLOCK_SIZE;
	aggregate_oblique_path_kernel<1, 1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, left, right, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void enqueue_aggregate_upright2downleft_path(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	static const unsigned int SUBGROUP_SIZE = MAX_DISPARITY / DP_BLOCK_SIZE;
	static const unsigned int PATHS_PER_BLOCK = BLOCK_SIZE / SUBGROUP_SIZE;

	const int gdim = (width + height + PATHS_PER_BLOCK - 2) / PATHS_PER_BLOCK;
	const int bdim = BLOCK_SIZE;
	aggregate_oblique_path_kernel<-1, 1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, left, right, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void enqueue_aggregate_downright2upleft_path(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	static const unsigned int SUBGROUP_SIZE = MAX_DISPARITY / DP_BLOCK_SIZE;
	static const unsigned int PATHS_PER_BLOCK = BLOCK_SIZE / SUBGROUP_SIZE;

	const int gdim = (width + height + PATHS_PER_BLOCK - 2) / PATHS_PER_BLOCK;
	const int bdim = BLOCK_SIZE;
	aggregate_oblique_path_kernel<-1, -1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, left, right, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void enqueue_aggregate_downleft2upright_path(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	static const unsigned int SUBGROUP_SIZE = MAX_DISPARITY / DP_BLOCK_SIZE;
	static const unsigned int PATHS_PER_BLOCK = BLOCK_SIZE / SUBGROUP_SIZE;

	const int gdim = (width + height + PATHS_PER_BLOCK - 2) / PATHS_PER_BLOCK;
	const int bdim = BLOCK_SIZE;
	aggregate_oblique_path_kernel<1, -1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, left, right, width, height, p1, p2, min_disp);
}


template void enqueue_aggregate_upleft2downright_path<64u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_upleft2downright_path<128u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_upleft2downright_path<256u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_upright2downleft_path<64u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_upright2downleft_path<128u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_upright2downleft_path<256u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downright2upleft_path<64u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downright2upleft_path<128u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downright2upleft_path<256u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downleft2upright_path<64u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downleft2upright_path<128u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void enqueue_aggregate_downleft2upright_path<256u>(
	cost_type *dest,
	const feature_type *left,
	const feature_type *right,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

}
}
